﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const* const func, const char* const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

void output_image(const float* fb, int nx, int ny  ,const char* fileName) 
{
    std::ofstream outfile;
    
    outfile.open(fileName, std::ios_base::app); // append instead of overwrit

    // Output FB as Image
    outfile << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j * 3 * nx + i * 3;
            float r = fb[pixel_index + 0];
            float g = fb[pixel_index + 1];
            float b = fb[pixel_index + 2];
            int ir = int(255.99 * r);
            int ig = int(255.99 * g);
            int ib = int(255.99 * b);
            outfile << ir << " " << ig << " " << ib << "\n";
        }
    }
}

__global__ void render(float* fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x * 3 + i * 3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}


int main() {
    //image 
    auto fileName = "C:\\Users\\Shyam Poovaiah\\Desktop\\image.ppm";

    // Image Dimensions
    const int nx = 256;
    const int ny = 256;

    

   

    

    int num_pixels = nx * ny;
    size_t fb_size = 3 * num_pixels * sizeof(float); //3 for rgb

    // allocate FB
    float* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    int tx = 8;
    int ty = 8;
   
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render <<<blocks, threads >>> (fb, nx, ny);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    output_image(fb, nx, ny, fileName);

    checkCudaErrors(hipFree(fb));

    return 0;
}






