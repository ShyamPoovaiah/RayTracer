#include "hip/hip_runtime.h"
﻿#include "rtweekend.h"
#include "color.h"
#include "hittable_list.h"
#include "sphere.h"
#include <iostream>
#include "utilities.h"
#include "camera.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

__device__ vec3 ray_color(const ray& r, hittable** world) {
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5f * vec3(rec.normal.x() + 1.0f, rec.normal.y() + 1.0f, rec.normal.z() + 1.0f);
    }
    else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f * (unit_direction.y() + 1.0f);
        return (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* pixels, int max_x, int max_y, int ns, camera** cam, hittable** world, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v);
        col += ray_color(r, world);
    }
    pixels[pixel_index] = col / float(ns);
}

__global__ void create_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list) = new sphere(vec3(0, 0, -1), 0.5);
        *(d_list + 1) = new sphere(vec3(0, -100.5, -1), 100);
        *d_world = new hittable_list(d_list, 2);
        *d_camera = new camera();
    }
}

__global__ void free_world(hittable** d_list, hittable** d_world, camera** d_camera) {
    delete* (d_list);
    delete* (d_list + 1);
    delete* d_world;
    delete* d_camera;
}



int main() {

    // Image

    const auto aspect_ratio = 16.0 / 9.0;
    const int nx = 400;
    const int ny = static_cast<int>(nx / aspect_ratio);
    const int samples_per_pixel = 100;
    auto num_pixels = nx * ny;
    auto pixel_size = num_pixels * sizeof(vec3);

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));


    std::cerr << "Creating world... \n";
    // make our world of hittables
    hittable** d_list;
    checkCudaErrors(hipMalloc((void**)&d_list, 2 * sizeof(hittable*)));
    hittable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hittable*)));
    camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(camera*)));
    create_world<<<1, 1 >>> (d_list, d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Camera

    auto viewport_height = 2.0;
    auto viewport_width = aspect_ratio * viewport_height;
    auto focal_length = 1.0;

    auto origin = point3(0, 0, 0);
    auto horizontal = vec3(viewport_width, 0, 0);
    auto vertical = vec3(0, viewport_height, 0);
    auto lower_left_corner = origin - horizontal / 2 - vertical / 2 - vec3(0, 0, focal_length);

    int tx = 8;
    int ty = 8;

    point3* pixels;
    
    checkCudaErrors(hipMallocManaged((void**)&pixels, pixel_size));

    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);

    render_init <<<blocks, threads >>> (nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    std::cerr << "Rendering... \n";
    render << <blocks, threads >> > (pixels, nx, ny, samples_per_pixel, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto fileName = "C:\\Users\\Shyam Poovaiah\\Desktop\\image.ppm";
    output_image(pixels, nx, ny, fileName);

    std::cerr << "Freeing world... \n";
    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world <<<1, 1 >>> (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(pixels));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
   

    std::cerr << "\nDone.\n";
}